#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#ifdef HAVE_MPI
#include <mpi.h>
#endif
#include "../devicehost.h"
#include "./private.h"

//FIX THIS: It's also in ../private.h
#define PRTHRESH 1000000L

extern "C" {

__host__ void GPTLprint_gpustats (int nwarps_found, // From gptlget_gpusizes
			 int nwarps_timed, // From gptlget_gpusizes
			 // From gptlget_overhead_gpu:
			 long long ftn_ohdgpu,            // Fortran wrapper overhead
			 long long get_thread_num_ohdgpu, /* Getting my thread index */
			 long long genhashidx_ohdgpu,     /* Generating hash index */
			 long long getentry_ohdgpu,       /* Finding entry in hash table */
			 long long utr_ohdgpu,            /* Underlying timing routine */
			 long long self_ohdgpu,           // Cost est. for timing this region
			 long long parent_ohdgpu,         // Cost est. to parent of this region
			 // From gptlfill_gpustats:
			 Gpustats gpustats[MAX_GPUTIMERS],
			 int max_name_len_gpu,
			 int ngputimers,
			 // From gptlget_memstats_gpu:
			 float hashmem,
			 float regionmem)
{
  FILE *fp;                    // output file for GPU stats
  int khz;                     // clock rate returned from GPTL_get_gpu_props
  int warpsize;                // warp size returned from GPTL_get_gpu_props
  double gpu_hz;               // khz turned into float hz
  int maxthreads_gpu; 
  int devnum;

  int count_max, count_min;
  int extraspace;
  int i, n;
  int ret;
  double wallmax, wallmin;
  double self, parent;
  double tot_ohdgpu;
  int myrank = 0;
  int mpi_active;
  char outfile[14];
#define HOSTSIZE 32
  char hostname[HOSTSIZE];
  static const char *thisfunc = "GPTLprint_gpustats";

#ifdef HAVE_MPI
  ret = MPI_Initialized (&mpi_active);
  if (mpi_active)
    ret = MPI_Comm_rank (MPI_COMM_WORLD, &myrank);
#endif

  ret = GPTLget_gpu_props (&khz, &warpsize, &devnum);
  gpu_hz = khz * 1000.;

  sprintf (outfile, "timing.%d", myrank);
  if ( ! (fp = fopen (outfile, "a")))
    fp = stderr;
  fprintf (fp, "\n\nGPU Results:\n");
  fprintf (fp, "%s: device number=%d\n", thisfunc, devnum);
  ret = gethostname (hostname, HOSTSIZE);
  fprintf (fp, "%s: hostname=%s\n", thisfunc, hostname);

  fprintf (fp, "Underlying timing routine was clock64()\n");
  tot_ohdgpu = (ftn_ohdgpu + get_thread_num_ohdgpu + genhashidx_ohdgpu + 
		getentry_ohdgpu + utr_ohdgpu) / gpu_hz;
  fprintf (fp, "Total overhead of 1 GPTLstart_gpu or GPTLstop_gpu call=%g seconds\n", tot_ohdgpu);
  fprintf (fp, "Components are as follows:\n");
  fprintf (fp, "Fortran layer:             %7.1e = %5.1f%% of total\n", 
	   ftn_ohdgpu / gpu_hz, ftn_ohdgpu * 100. / (tot_ohdgpu * gpu_hz) );
  fprintf (fp, "Get thread number:         %7.1e = %5.1f%% of total\n", 
	   get_thread_num_ohdgpu / gpu_hz, get_thread_num_ohdgpu * 100. / (tot_ohdgpu * gpu_hz) );
  fprintf (fp, "Generate hash index:       %7.1e = %5.1f%% of total\n", 
	   genhashidx_ohdgpu / gpu_hz, genhashidx_ohdgpu * 100. / (tot_ohdgpu * gpu_hz) );
  fprintf (fp, "Find hashtable entry:      %7.1e = %5.1f%% of total\n", 
	   getentry_ohdgpu / gpu_hz, getentry_ohdgpu * 100. / (tot_ohdgpu * gpu_hz) );
  fprintf (fp, "Underlying timing routine: %7.1e = %5.1f%% of total\n", 
	   utr_ohdgpu / gpu_hz, utr_ohdgpu * 100. / (tot_ohdgpu * gpu_hz) );
  fprintf (fp, "\nGPU timing stats\n");
  fprintf (fp, "GPTL could handle up to %d warps (%d threads)\n", 
	   maxthreads_gpu / warpsize, maxthreads_gpu);
  fprintf (fp, "This setting can be changed with: GPTLsetoption(GPTLmaxthreads_gpu,<number>)\n");
  fprintf (fp, "%d warps were found\n", nwarps_found);
  fprintf (fp, "%d warps were timed\n", nwarps_timed);
  fprintf (fp, "Only warps which were timed are counted in the following stats\n");
  fprintf (fp, "Overhead estimates self_OH and parent_OH are for warp with \'maxcount\' calls\n");
  fprintf (fp, "OHD estimate assumes Fortran, and non-handle routines used\n");
  fprintf (fp, "Actual overhead can be reduced by using \'handle\' routines and \'_c\' Fortran routines\n");
  // Print header, padding to length of longest name
  extraspace = max_name_len_gpu - 4; // "name" is 4 chars
  for (i = 0; i < extraspace; ++i)
    fprintf (fp, " ");
  fprintf (fp, "name calls warps  wallmax (warp) wallmin (warp) maxcount (warp) mincount (warp) self_OH parent_OH\n");
  for (n = 0; n < ngputimers; ++n) {
    extraspace = max_name_len_gpu - strlen (gpustats[n].name);
    for (i = 0; i < extraspace; ++i)
      fprintf (fp, " ");
    fprintf (fp, "%s ", gpustats[n].name);             // regopm name
    fprintf (fp, "%5d ", gpustats[n].count);           // # start/stops of region 
    fprintf (fp, "%5d ", gpustats[n].nwarps);          // nwarps_timed involving name
    
    wallmax = gpustats[n].accum_max / gpu_hz;          // max time for name across warps
    if (wallmax < 0.01)
      fprintf (fp, "%8.2e ", wallmax);
    else
      fprintf (fp, "%8.3f ", wallmax);
    fprintf (fp, "%5d ",gpustats[n].accum_max_warp);   // warp number for max
    
    wallmin = gpustats[n].accum_min / gpu_hz;          // min time for name across warps
    if (wallmin < 0.01)
      fprintf (fp, "%8.2e ", wallmin);
    else
      fprintf (fp, "%8.3f ", wallmin);	       
    fprintf (fp, "%5d ",gpustats[n].accum_min_warp);   // warp number for min
    
    count_max = gpustats[n].count_max;
    if (count_max < PRTHRESH)
      fprintf (fp, "%9lu ", count_max);                // max count for region "name"
    else
      fprintf (fp, "%9.1e ", (float) count_max);
    fprintf (fp, "%5d ",gpustats[n].count_max_warp);   // warp which accounted for max times
    
    count_min = gpustats[n].count_min;                
    if (count_min < PRTHRESH)
      fprintf (fp, "%9lu ", count_min);                // min count for region "name"
    else
      fprintf (fp, "%9.1e ", (float) count_min);
    fprintf (fp, "%5d ",gpustats[n].count_min_warp);   // warp which accounted for max times

    self = gpustats[n].count_max * self_ohdgpu / gpu_hz;     // self ohd est
    if (self < 0.01)
      fprintf (fp, "%8.2e  ", self);
    else
      fprintf (fp, "%8.3f  ", self);	       
    
    parent = gpustats[n].count_max * parent_ohdgpu / gpu_hz; // parent ohd est
    if (self < 0.01)
      fprintf (fp, "%8.2e ", parent);
    else
      fprintf (fp, "%8.3f ", parent);	       
    
    fprintf (fp, "\n");
  }
  
  fprintf (fp, "\n");
  fprintf (fp, "Total GPTL GPU memory usage = %g KB\n", (hashmem + regionmem)*.001);
  fprintf (fp, "Components:\n");
  fprintf (fp, "Hashmem                     = %g KB\n" 
               "Regionmem                   = %g KB\n", hashmem*.001, regionmem*.001);
}

}
