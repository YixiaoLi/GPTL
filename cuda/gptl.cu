#include "hip/hip_runtime.h"
/*
** gptl.c
** Author: Jim Rosinski
**
** Main file contains most user-accessible GPTL functions
*/

#include <stdlib.h>        /* malloc */
#include <stdio.h>
#include <string.h>        /* memcpy */

#include "./private.h"
#include "./gptl.h"

__device__ static Timer **timers = 0;             /* linked list of timers */
__device__ static Timer **last = 0;               /* last element in list */
__device__ static int *max_depth;                 /* maximum indentation level encountered */
__device__ static int *max_name_len;              /* max length of timer name */
__device__ static int nthreads = -1;                /* num threads. Init to bad value */
__device__ static int maxthreads = -1;              /* max threads */
__device__ static bool disabled = false;          /* Timers disabled? */
__device__ static bool initialized = false;       /* GPTLinitialize has been called */
__device__ static bool verbose = false;           /* output verbosity */
__device__ static bool imperfect_nest;                 /* e.g. start(A),start(B),stop(A) */

/* Options, print strings, and default enable flags */
__device__ static Hashentry **hashtable;    /* table of entries */
__device__ static Timer ***callstack;       /* call stack */
__device__ static int *stackidx;            /* index into callstack: */
__device__ static int tablesize;
__device__ static int tablesizem1;
__device__ static int maxtimers;            /* max number of timers to pass back to CPU */

extern "C" {

/* Local function prototypes */
__device__ static inline int get_warp_num (void);         /* get 0-based thread number */
__device__ static inline unsigned int genhashidx (const char *);
__device__ static inline Timer *getentry (const Hashentry *, const char *, unsigned int);
__device__ static inline int update_parent_info (Timer *, Timer **, int);
__device__ static inline int update_stats (Timer *, const long long, const int);
__device__ static int update_ll_hash (Timer *, int, unsigned int);
__device__ static inline int update_ptr (Timer *, const int);
__device__ static inline int my_strlen (const char *);
__device__ static inline char *my_strcpy (char *, const char *);
__device__ static inline int my_strcmp (const char *, const char *);

/* These are invoked only from gptl.c */
__device__ extern int GPTLinitialize_gpu (const int, const int, const int, const int);
__device__ extern int GPTLenable_gpu (void);
__device__ extern int GPTLdisable_gpu (void);
__device__ extern int GPTLreset_gpu (void);

/* VERBOSE is a debugging ifdef local to the rest of this file */
#undef VERBOSE

/*
** GPTLinitialize_gpu (): Initialization routine must be called from single-threaded
**   region before any other timing routines may be called.  The need for this
**   routine could be eliminated if not targetting timing library for threaded
**   capability. 
** return value: 0 (success) or GPTLerror (failure)
*/
__device__ int GPTLinitialize_gpu (const int verbose_in,
				   const int tablesize_in,
				   const int maxthreads_in,
				   const int maxtimers_in)
{
  int i;                 /* loop index */
  int w;                 /* warp index */
  long long t1, t2;      /* returned from underlying timer */
  static const char *thisfunc = "GPTLinitialize_gpu";

  if (initialized)
    printf ("%s: has already been called\n", thisfunc);
  

  // Set global vars from input args
  verbose     = verbose_in;
  maxthreads    = maxthreads_in;
  tablesize   = tablesize_in;
  tablesizem1 = tablesize_in - 1;
  maxtimers   = maxtimers_in;

  /* Allocate space for global arrays */
  printf ("Calling GPTLallocate (gpu) for callstack\n");
  callstack     = (Timer ***)    GPTLallocate (maxthreads * sizeof (Timer **), thisfunc);
  printf ("Calling GPTLallocate (gpu) for stackidx\n");
  stackidx      = (int *)        GPTLallocate (maxthreads * sizeof (int), thisfunc);
  timers        = (Timer **)     GPTLallocate (maxthreads * sizeof (Timer *), thisfunc);
  last          = (Timer **)     GPTLallocate (maxthreads * sizeof (Timer *), thisfunc);
  max_depth     = (int *)        GPTLallocate (maxthreads * sizeof (int), thisfunc);
  max_name_len  = (int *)        GPTLallocate (maxthreads * sizeof (int), thisfunc);
  hashtable     = (Hashentry **) GPTLallocate (maxthreads * sizeof (Hashentry *), thisfunc);

  /* Initialize array values */
  for (w = 0; w < maxthreads; w++) {
    max_depth[w]    = -1;
    max_name_len[w] = 0;
    callstack[w] = (Timer **) GPTLallocate (MAX_STACK * sizeof (Timer *), thisfunc);
    hashtable[w] = (Hashentry *) GPTLallocate (tablesize * sizeof (Hashentry), thisfunc);
    for (i = 0; i < tablesize; i++) {
      hashtable[w][i].nument = 0;
    }

    /* Make a timer "GPTL_ROOT" to ensure no orphans, and to simplify printing. */
    timers[w] = (Timer *) GPTLallocate (sizeof (Timer), thisfunc);
    memset (timers[w], 0, sizeof (Timer));
    (void) my_strcpy (timers[w]->name, "GPTL_ROOT");
    timers[w]->onflg = true;
    last[w] = timers[w];

    stackidx[w] = 0;
    callstack[w][0] = timers[w];
    for (i = 1; i < MAX_STACK; i++)
      callstack[w][i] = 0;
  }

  if (verbose) {
    t1 = clock64 ();
    t2 = clock64 ();
    if (t1 > t2)
      printf ("%s: negative delta-t=%lld\n", thisfunc, t2-t1);

    printf ("Per call overhead est. t2-t1=%g should be near zero\n", t2-t1);
    printf ("Underlying wallclock timing routine is clock64\n");
  }

  imperfect_nest = false;
  initialized = true;
  return 0;
}

/*
** GPTLfinalize_gpu (): Finalization routine must be called from single-threaded
**   region. Free all malloc'd space
**
** return value: 0 (success) or GPTLerror (failure)
*/
__device__ int GPTLfinalize_gpu (void)
{
  int w;                /* warp index */
  Timer *ptr, *ptrnext; /* ll indices */
  static const char *thisfunc = "GPTLfinalize_gpu";

  if ( ! initialized)
    return GPTLerror_1s ("%s: initialization was not completed\n", thisfunc);

  for (w = 0; w < maxthreads; ++w) {
    free (hashtable[w]);
    hashtable[w] = NULL;
    free (callstack[w]);
    for (ptr = timers[w]; ptr; ptr = ptrnext) {
      ptrnext = ptr->next;
      if (ptr->nparent > 0) {
        free (ptr->parent);
        free (ptr->parent_count);
      }
      if (ptr->nchildren > 0)
        free (ptr->children);
      free (ptr);
    }
  }

  free (callstack);
  free (stackidx);
  free (timers);
  free (last);
  free (max_depth);
  free (max_name_len);
  free (hashtable);

  GPTLreset_errors ();

  /* Reset initial values */
  timers = 0;
  last = 0;
  max_depth = 0;
  max_name_len = 0;
  nthreads = -1;
  maxthreads = -1;
  disabled = false;
  initialized = false;
  verbose = false;

  return 0;
}

/*
** GPTLstart: start a timer
**
** Input arguments:
**   name: timer name
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ int GPTLstart_gpu (const char *name)               /* timer name */
{
  Timer *ptr;        /* linked list pointer */
  int w;             /* warp index (of this thread) */
  int numchars;      /* number of characters to copy */
  unsigned int indx; /* hash table index */
  static const char *thisfunc = "GPTLstart";

  if (disabled)
    return 0;

  if ( ! initialized)
    return GPTLerror_2s ("%s name=%s: GPTLinitialize has not been called\n", thisfunc, name);

  if ((w = get_warp_num ()) == -1)
    return GPTLerror_1s ("%s: bad return from get_warp_num\n", thisfunc);

  // Return if not thread 0 of the warp
  if (w < 0)
    return 0;

  /* ptr will point to the requested timer in the current list, or NULL if this is a new entry */
  indx = genhashidx (name);
  ptr = getentry (hashtable[w], name, indx);

  /* 
  ** Recursion => increment depth in recursion and return.  We need to return 
  ** because we don't want to restart the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (ptr && ptr->onflg) {
    ++ptr->recurselvl;
    return 0;
  }

  /*
  ** Increment stackidx[w] unconditionally. This is necessary to ensure the correct
  ** behavior when GPTLstop decrements stackidx[w] unconditionally.
  */
  if (++stackidx[w] > MAX_STACK-1)
    return GPTLerror_1s ("%s: stack too big\n", thisfunc);

  if ( ! ptr) { /* Add a new entry and initialize */
    ptr = (Timer *) GPTLallocate (sizeof (Timer), thisfunc);
    memset (ptr, 0, sizeof (Timer));

    numchars = MIN (my_strlen (name), MAX_CHARS);
    memcpy (ptr->name, name, numchars);
    ptr->name[numchars] = '\0';

    if (update_ll_hash (ptr, w, indx) != 0)
      return GPTLerror_1s ("%s: update_ll_hash error\n", thisfunc);
  }

  if (update_parent_info (ptr, callstack[w], stackidx[w]) != 0)
    return GPTLerror_1s ("%s: update_parent_info error\n", thisfunc);

  if (update_ptr (ptr, w) != 0)
    return GPTLerror_1s ("%s: update_ptr error\n", thisfunc);

  return (0);
}

/*
** GPTLinit_handle: Initialize a handle for further use by GPTLstart_handle() and GPTLstop_handle()
**
** Input arguments:
**   name: timer name
**
** Output arguments:
**   handle: hash value corresponding to "name"
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ int GPTLinit_handle_gpu (const char *name,     /* timer name */
				int *handle)          /* handle (output if input value is zero) */
{
  if (disabled)
    return 0;

  *handle = (int) genhashidx (name);
  return 0;
}

/*
** GPTLstart_handle: start a timer based on a handle
**
** Input arguments:
**   name: timer name (required when on input, handle=0)
**   handle: pointer to timer matching "name"
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ int GPTLstart_handle_gpu (const char *name,  /* timer name */
				 int *handle)       /* handle (output if input value is zero) */
{
  Timer *ptr;        /* linked list pointer */
  int w;             /* warp index (of this thread) */
  int numchars;      /* number of characters to copy */
  static const char *thisfunc = "GPTLstart_handle";

  if (disabled)
    return 0;

  if ( ! initialized)
    return GPTLerror_2s ("%s name=%s: GPTLinitialize has not been called\n", thisfunc, name);

  if ((w = get_warp_num ()) == -1)
    return GPTLerror_1s ("%s: bad return from get_warp_num\n", thisfunc);

  // Return if not thread 0 of the warp
  if (w < 0)
    return 0;

  /*
  ** If handle is zero on input, generate the hash entry and return it to the user.
  ** Otherwise assume it's a previously generated hash index passed in by the user.
  ** Don't need a critical section here--worst case multiple threads will generate the
  ** same handle and store to the same memory location, and this will only happen once.
  */
  if (*handle == 0) {
    *handle = (int) genhashidx (name);
#ifdef VERBOSE
    printf ("%s: name=%s warp %d generated handle=%d\n", thisfunc, name, w, *handle);
#endif
  } else if ((unsigned int) *handle > tablesizem1) {
    return GPTLerror_1s2d ("%s: Bad input handle=%u exceeds tablesizem1=%d\n", 
			   thisfunc, *handle, tablesizem1);
  }

  ptr = getentry (hashtable[w], name, (unsigned int) *handle);
  
  /* 
  ** Recursion => increment depth in recursion and return.  We need to return 
  ** because we don't want to restart the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (ptr && ptr->onflg) {
    ++ptr->recurselvl;
    return 0;
  }

  /*
  ** Increment stackidx[w] unconditionally. This is necessary to ensure the correct
  ** behavior when GPTLstop decrements stackidx[w] unconditionally.
  */
  if (++stackidx[w] > MAX_STACK-1)
    return GPTLerror_1s ("%s: stack too big\n", thisfunc);

  if ( ! ptr) { /* Add a new entry and initialize */
    ptr = (Timer *) GPTLallocate (sizeof (Timer), thisfunc);
    memset (ptr, 0, sizeof (Timer));

    numchars = MIN (my_strlen (name), MAX_CHARS);
    memcpy (ptr->name, name, numchars);
    ptr->name[numchars] = '\0';

    if (update_ll_hash (ptr, w, (unsigned int) *handle) != 0)
      return GPTLerror_1s ("%s: update_ll_hash error\n", thisfunc);
  }

  if (update_parent_info (ptr, callstack[w], stackidx[w]) != 0)
    return GPTLerror_1s ("%s: update_parent_info error\n", thisfunc);

  if (update_ptr (ptr, w) != 0)
    return GPTLerror_1s ("%s: update_ptr error\n", thisfunc);

  return (0);
}

/*
** update_ll_hash: Update linked list and hash table.
**                 Called by all GPTLstart* routines when there is a new entry
**
** Input arguments:
**   ptr:  pointer to timer
**   w:    warp index
**   indx: hash index
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ static int update_ll_hash (Timer *ptr, int w, unsigned int indx)
{
  int nchars;      /* number of chars */
  int nument;      /* number of entries */
  static const char *thisfunc = "update_ll_hash";

  nchars = my_strlen (ptr->name);
  if (nchars > max_name_len[w])
    max_name_len[w] = nchars;

  last[w]->next = ptr;
  last[w] = ptr;
  if (hashtable[w][indx].nument > MAXENT-1)
    return GPTLerror_2s ("%s: %s has too many hash collisions\n", thisfunc, ptr->name);

  ++hashtable[w][indx].nument;
  nument = hashtable[w][indx].nument;
  hashtable[w][indx].entries[nument-1] = ptr;
  return 0;
}

/*
** update_ptr: Update timer contents. Called by GPTLstart and GPTLstart_handle
**
** Input arguments:
**   ptr:  pointer to timer
**   w:    warp index
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ static inline int update_ptr (Timer *ptr, const int w)
{
  long long tp2;    /* time stamp */

  ptr->onflg = true;
  tp2 = clock64 ();
  ptr->wall.last = tp2;
  return 0;
}

/*
** update_parent_info: update info about parent, and in the parent about this child
**                     Called by all GPTLstart* routines
**
** Arguments:
**   ptr:  pointer to timer
**   callstackt: callstack for this warp
**   stackidxt:  stack index for this warp
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ static inline int update_parent_info (Timer *ptr, 
						 Timer **callstackt, 
						 int stackidxt) 
{
  int n;             /* loop index through known parents */
  Timer *pptr;       /* pointer to parent in callstack */
  int nparent;       /* number of parents */
  static const char *thisfunc = "update_parent_info";

  if ( ! ptr )
    return -1;

  if (stackidxt < 0)
    return GPTLerror_1s ("%s: called with negative stackidx\n", thisfunc);

  callstackt[stackidxt] = ptr;

  /* Bump orphan count if the region has no parent (should never happen since "GPTL_ROOT" added) */
  if (stackidxt == 0) {
    ++ptr->norphan;
    return 0;
  }

  pptr = callstackt[stackidxt-1];

  /* If this parent occurred before, bump its count */
  for (n = 0; n < ptr->nparent; ++n) {
    if (ptr->parent[n] == pptr) {
      ++ptr->parent_count[n];
      break;
    }
  }

  /* If this is a new parent, update info */
  if (n == ptr->nparent) {
    if (ptr->nparent == MAXPARENT)
      return GPTLerror_2s1d ("%s: cannot add newparent=%s because it exceeds MAXPARENT=%d\n",
			     thisfunc, pptr->name, MAXPARENT);
    ++ptr->nparent;
    nparent = ptr->nparent;

    ptr->parent[nparent-1] = pptr;
    ptr->parent_count[nparent-1] = 1;
  }
  return 0;
}

/*
** GPTLstop: stop a timer
**
** Input arguments:
**   name: timer name
**
** Return value: 0 (success) or -1 (failure)
*/
__device__ int GPTLstop_gpu (const char *name)               /* timer name */
{
  long long tp1 = 0;         /* time stamp */
  Timer *ptr;                /* linked list pointer */
  int w;                     /* warp number for this process */
  unsigned int indx;         /* index into hash table */
  static const char *thisfunc = "GPTLstop";

  if (disabled)
    return 0;

  if ( ! initialized)
    return GPTLerror_1s ("%s: GPTLinitialize has not been called\n", thisfunc);

  /* Get the timestamp */
    
  tp1 = clock64 ();

  if ((w = get_warp_num ()) == -1)
    return GPTLerror_1s ("%s: bad return from get_warp_num\n", thisfunc);

  // Return if not thread 0 of the warp
  if (w < 0)
    return 0;

  indx = genhashidx (name);
  if (! (ptr = getentry (hashtable[w], name, indx)))
    return GPTLerror_1s1d1s ("%s warp %d: timer for %s had not been started.\n", thisfunc, w, name);

  if ( ! ptr->onflg )
    return GPTLerror_2s ("%s: timer %s was already off.\n", thisfunc, ptr->name);

  ++ptr->count;

  /* 
  ** Recursion => decrement depth in recursion and return.  We need to return
  ** because we don't want to stop the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (ptr->recurselvl > 0) {
    --ptr->recurselvl;
    return 0;
  }

  if (update_stats (ptr, tp1, w) != 0)
    return GPTLerror_1s ("%s: error from update_stats\n", thisfunc);

  return 0;
}

/*
** GPTLstop_handle: stop a timer based on a handle
**
** Input arguments:
**   name: timer name (used only for diagnostics)
**   handle: pointer to timer
**
** Return value: 0 (success) or -1 (failure)
*/
__device__ int GPTLstop_handle_gpu (const char *name,     /* timer name */
				const int *handle)    /* handle */
{
  long long tp1 = 0;         /* time stamp */
  Timer *ptr;                /* linked list pointer */
  int w;                     /* warp number for this process */
  unsigned int indx;
  static const char *thisfunc = "GPTLstop_handle";

  if (disabled)
    return 0;

  if ( ! initialized)
    return GPTLerror_1s ("%s: GPTLinitialize has not been called\n", thisfunc);

  /* Get the timestamp */
  tp1 = clock64 ();

  if ((w = get_warp_num ()) == -1)
    return GPTLerror_1s ("%s: bad return from get_warp_num\n", thisfunc);

  // Return if not thread 0 of the warp
  if (w < 0)
    return 0;

  indx = (unsigned int) *handle;
  if (indx == 0 || indx > tablesizem1)
    return GPTLerror_1s1d1s ("%s: bad input handle=%u for timer %s.\n", thisfunc, (int) indx, name);
  
  if ( ! (ptr = getentry (hashtable[w], name, indx)))
    return GPTLerror_1s1d1s ("%s: handle=%u has not been set for timer %s.\n", 
			     thisfunc, (int) indx, name);

  if ( ! ptr->onflg )
    return GPTLerror_2s ("%s: timer %s was already off.\n", thisfunc, ptr->name);

  ++ptr->count;

  /* 
  ** Recursion => decrement depth in recursion and return.  We need to return
  ** because we don't want to stop the timer.  We want the reported time for
  ** the timer to reflect the outermost layer of recursion.
  */
  if (ptr->recurselvl > 0) {
    --ptr->recurselvl;
    return 0;
  }

  if (update_stats (ptr, tp1, w) != 0)
    return GPTLerror_1s ("%s: error from update_stats\n", thisfunc);

  return 0;
}

/*
** update_stats: update stats inside ptr. Called by GPTLstop, GPTLstop_handle
**
** Input arguments:
**   ptr: pointer to timer
**   tp1: input time stamp
**   w: warp index
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ static inline int update_stats (Timer *ptr, 
					   const long long tp1, 
					   const int w)
{
  long long delta;   /* difference */
  int bidx;          /* bottom of call stack */
  Timer *bptr;       /* pointer to last entry in call stack */
  static const char *thisfunc = "update_stats";

  ptr->onflg = false;

  delta = tp1 - ptr->wall.last;
  ptr->wall.accum += delta;

  if (delta < 0)
    printf ("GPTL: %s: negative delta=%lld\n", thisfunc, delta);

  if (ptr->count == 1) {
    ptr->wall.max = delta;
    ptr->wall.min = delta;
  } else {
    if (delta > ptr->wall.max)
      ptr->wall.max = delta;
    if (delta < ptr->wall.min)
      ptr->wall.min = delta;
  }

  /* Verify that the timer being stopped is at the bottom of the call stack */
  bidx = stackidx[w];
  bptr = callstack[w][bidx];
  if (ptr != bptr) {
    imperfect_nest = true;
    printf ("%s: Got timer=%s expected btm of call stack=%s\n", 
	    thisfunc, ptr->name, bptr->name);
  }

  --stackidx[w];           /* Pop the callstack */
  if (stackidx[w] < -1) {
    stackidx[w] = -1;
    return GPTLerror_1s ("%s: tree depth has become negative.\n", thisfunc);
  }

  return 0;
}

/*
** GPTLenable_gpu: enable timers
**
** Return value: 0 (success)
*/
  __device__ int GPTLenable_gpu (void)
{
  disabled = false;
  return (0);
}

/*
** GPTLdisable_gpu: disable timers
**
** Return value: 0 (success)
*/
__device__ int GPTLdisable_gpu (void)
{
  disabled = true;
  return (0);
}

/*
** GPTLreset_gpu: reset all timers to 0
**
** Return value: 0 (success) or GPTLerror (failure)
*/
__device__ int GPTLreset_gpu (void)
{
  int w;             /* index over threads */
  Timer *ptr;        /* linked list index */
  static const char *thisfunc = "GPTLreset";

  if ( ! initialized)
    return GPTLerror_1s ("%s: GPTLinitialize has not been called\n", thisfunc);

  for (w = 0; w < nthreads; w++) {
    for (ptr = timers[w]; ptr; ptr = ptr->next) {
      ptr->onflg = false;
      ptr->count = 0;
      memset (&ptr->wall, 0, sizeof (ptr->wall));
    }
  }

  if (verbose)
    printf ("%s: accumulators for all GPU timers set to zero\n", thisfunc);

  return 0;
}

/*
** genhashidx: generate hash index
**
** Input args:
**   name: string to be hashed on
**
** Return value: hash value
*/
#define NEWWAY
__device__ static inline unsigned int genhashidx (const char *name)
{
  const unsigned char *c;       /* pointer to elements of "name" */
  unsigned int indx;            /* return value of function */
#ifdef NEWWAY
  unsigned int mididx, lastidx; /* mid and final index of name */

  lastidx = my_strlen (name) - 1;
  mididx = lastidx / 2;
#else
  int i;                        /* iterator (OLDWAY only) */
#endif
  /* 
  ** Disallow a hash index of zero (by adding 1 at the end) since user input of an uninitialized 
  ** value, though an error, has a likelihood to be zero.
  */
#ifdef NEWWAY
  c = (unsigned char *) name;
  indx = (MAX_CHARS*c[0] + (MAX_CHARS-mididx)*c[mididx] + (MAX_CHARS-lastidx)*c[lastidx]) % tablesizem1 + 1;
#else
  indx = 0;
  i = MAX_CHARS;
#pragma unroll(2)
  for (c = (unsigned char *) name; *c && i > 0; ++c) {
    indx += i*(*c);
    --i;
  }
  indx = indx % tablesizem1 + 1;
#endif

  return indx;
}

/*
** getentry: find the entry in the hash table and return a pointer to it.
**
** Input args:
**   hashtable: the hashtable (array)
**   indx:      hashtable index
**
** Return value: pointer to the entry, or NULL if not found
*/
__device__ static inline Timer *getentry (const Hashentry *hashtable, /* hash table */
					  const char *name,           /* name to hash */
					  unsigned int indx)          /* hash index */
{
  int i;                      /* loop index */
  Timer *ptr = 0;             /* return value when entry not found */

  /* 
  ** If nument exceeds 1 there was one or more hash collisions and we must search
  ** linearly through the array of names with the same hash for a match
  */
  for (i = 0; i < hashtable[indx].nument; i++) {
    if (STRMATCH (name, hashtable[indx].entries[i]->name)) {
      ptr = hashtable[indx].entries[i];
      break;
    }
  }
  return ptr;
}

/*
** placebo: does nothing and returns zero always. Useful for estimating overhead costs
*/
__device__ static int init_placebo ()
{
  return 0;
}

__device__ static inline long long utr_placebo ()
{
  return (long long) 0;
}

__device__ static inline int get_warp_num ()
{
  static const char *thisfunc = "get_warp_num";
  int warpId;
  int blockId = blockIdx.x 
    + blockIdx.y * gridDim.x 
    + gridDim.x * gridDim.y * blockIdx.z; 
  int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
    + (threadIdx.z * (blockDim.x * blockDim.y))
    + (threadIdx.y * blockDim.x)
    + threadIdx.x;

#ifdef USE_WARP_NUM
  if (threadId % WARPSIZE != 0)
    return NOT_ROOT_OF_WARP;

  warpId = threadId / WARPSIZE;
#else
  warpId = threadId;
#endif
  if (warpId > maxthreads-1)
    return GPTLerror_1s2d ("%s: warpId=%d exceeds maxthreads=%d\n", thisfunc, warpId, maxthreads);
  else
    return warpId;
}

//JR want to use parameters to dimension arrays but nvcc is not C99 compliant
__device__ int GPTLfill_gpustats (Gpustats gpustats[][DEFAULT_MAXTIMERS_GPU], 
                                  int *max_name_len_out)
{
  int w;      // warp index
  int n;      // timer index
  int p;      // parent index
  Timer *ptr; // loop through linked list
  static const char *thisfunc = "GPTLfill_gpustats";

  for (w = 0; w < nthreads; ++w) {
    for (ptr = timers[w], n = 0; ptr; ptr = ptr->next, ++n) {
      if (n > maxtimers-1) 
	return GPTLerror_1s1d ("%s: Truncating timer output at %d entries."
			       "Suggest increase maxtimers array size to fill_output_stats()\n",
			       thisfunc, maxtimers);
      (void) my_strcpy (gpustats[w][n].name, ptr->name);
      gpustats[w][n].count       = ptr->count;
      gpustats[w][n].accum       = ptr->wall.accum;
      gpustats[w][n].max         = ptr->wall.max;
      gpustats[w][n].min         = ptr->wall.min;
      gpustats[w][n].nparent     = ptr->nparent;
      for (p = 0; p < ptr->nparent; ++p) {
	if (p > MAXPARENT-1) {
	  printf ("%s w=%d Timer=%s: Truncating parent count at MAXPARENT=%d out of %d\n", 
		  thisfunc, w, ptr->name, MAXPARENT, ptr->nparent);
	  break;
	}
	(void) my_strcpy (gpustats[w][n].parentname[p], ptr->parent[p]->name);
      }
    }
    max_name_len_out[w] = max_name_len[w];
  }
  return 0;
}

__device__ static inline int my_strlen (const char *str)
{
  int i;

  for (i = 0; str[i] != '\0'; ++i);
  return i;
}

__device__ static inline char *my_strcpy (char *dest, const char *src)
{
  char *ret = dest;

  while (*src != '\0')
    *dest++ = *src++;
  *dest = '\0';
  return ret;
}

__device__ static inline int my_strcmp (const char *str1, const char *str2)
{
  while (*str1 == *str2) {
    if (*str1 == '\0')
      break;
    ++str1;
    ++str2;
  }
  return (int) (*str1 - *str2);
}

}
