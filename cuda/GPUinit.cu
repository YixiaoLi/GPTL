// Routine to initialize the GPU
// Author:  Jacques Middlecoff
// Date:  September 2010 
// May 2014 Removed CUT_DEVICE_INIT - Middlecoff
// For Fortran this routine does nothing except return error=0.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern "C" void gpuinit_ (int *npes,int *me,int *max_accelerators_per_node,int *GPUrun,int *error) {

hipDeviceProp_t deviceProp;

*GPUrun = 1;
*error = 0;

#if CUDART_VERSION < 2020
#error "This CUDART version does not support mapped memory!\n"
#endif

// Get properties and verify device 0 supports mapped memory
*error = hipGetDeviceProperties(&deviceProp, 0);
if(*error != hipSuccess) {
  printf("GPUinit.cu: cudaGetDeviceProperties error %d \n",*error);
  printf("cudaSuccess,cudaErrorInvalidDevice,cudaErrorDeviceAlreadyInUse %d %d %d \n",hipSuccess,hipErrorInvalidDevice,hipErrorContextAlreadyInUse);
  return;
}
if(!deviceProp.canMapHostMemory) {
  printf("GPUinit.cu: Device %d cannot map host memory!\n", 0);
  *error = -88;
  return;
}

*error = hipSetDevice(*me%*max_accelerators_per_node);
if(*error != hipSuccess) {
  printf("GPUinit.cu: cudaSetDeviceProperties error %d %d %d \n",*error,*me,*max_accelerators_per_node);
  printf("cudaSuccess, %d \n",hipSuccess);
  return;
}
printf("MPI rank %d on GPU %d \n",*me,*me%*max_accelerators_per_node);

// set the device flags for mapping host memory
//*error = cudaSetDeviceFlags(cudaDeviceMapHost);
//if(*error != cudaSuccess) {
//  printf("GPUinit.cu: cudaSetDeviceFlags error %d \n",*error);
//  return;
//}
}



